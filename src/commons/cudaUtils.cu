#include "cudaUtils.h"
//int cudaErr(hipError_t e,const char* errStr);
int spMatCpyCSR(spmat* m,spmat* dst){
	char* errS = "spMatCpyCSR";
	//prepare the dst-copy struct
	spmat dstLocal;
	memset(&dstLocal,0,sizeof(dstLocal));
	dstLocal.M  = m->M;
	dstLocal.N  = m->N;
	dstLocal.NZ = m->NZ;

	ulong nz = m->NZ;
	//allocs and copies for the CSR fields inside @m
	if(cudaErr( hipMalloc( &(dstLocal.JA),sizeof(*(dstLocal.JA))*nz), errS))					goto err;
	if(cudaErr( hipMemcpy( dstLocal.JA,m->JA,sizeof(*(dstLocal.JA))*nz,dirUp), errS))			goto err;

	if(cudaErr( hipMalloc( &(dstLocal.AS),sizeof(*(dstLocal.AS)) * nz ), errS))				goto err;
	if(cudaErr( hipMemcpy( dstLocal.AS,m->AS,sizeof(*(dstLocal.AS))*nz,dirUp), errS))			goto err;
	
	if(cudaErr( hipMalloc( &(dstLocal.IRP),sizeof(*(dstLocal.IRP)) * (dstLocal.M+1) ), errS))		  	goto err;
	if(cudaErr( hipMemcpy( dstLocal.IRP,m->IRP,sizeof(*(dstLocal.IRP)) * (dstLocal.M+1),dirUp), errS)) goto err;
	#ifdef ROWLENS
	if(cudaErr( hipMalloc( &(dstLocal.RL),sizeof(*(dstLocal.RL)) * dstLocal.M ), errS))			goto err;
	if(cudaErr( hipMemcpy( dstLocal.RL,m->RL,sizeof(*(dstLocal.RL)) * dstLocal.M,dirUp), errS))	goto err;
	#endif
	//write dst-copy in destination CUDA mem struct
	if(cudaErr( hipMemcpy( dst,&dstLocal,sizeof(*dst),dirUp), errS))	goto err;
	return EXIT_SUCCESS;

	err:
	hipFree(dstLocal.JA);
	hipFree(dstLocal.AS);
	hipFree(dstLocal.IRP);
	#ifdef ROWLENS
	hipFree(dstLocal.RL);
	#endif
	return EXIT_FAILURE;
}
int spMatCpyELL(spmat* m,spmat* dst,size_t* pitchJA,size_t* pitchAS){
	char* errS = "spMatCpyELL";
	//prepare the dst-copy struct
	spmat dstLocal;
	memset(&dstLocal,0,sizeof(dstLocal));
	dstLocal.M = m->M;
	dstLocal.N = m->N;
	dstLocal.NZ= m->NZ;
	dstLocal.MAX_ROW_NZ= m->MAX_ROW_NZ;
	
	ulong nz = m->NZ;
	ulong maxRow = m->MAX_ROW_NZ;
	//allocs and copies for the ELL fields inside @m
	if(cudaErr( hipMallocPitch(&(dstLocal.JA),pitchJA,maxRow,m->M), errS))						goto err;
	if(cudaErr( hipMemcpy2D(dstLocal.JA,*pitchJA,m->JA,maxRow,maxRow,m->M,dirUp),errS))		goto err;
	
  	if(cudaErr( hipMallocPitch(&(dstLocal.AS),pitchAS,maxRow,m->M), errS))						goto err;
	if(cudaErr( hipMemcpy2D(dstLocal.AS,*pitchAS,m->AS,maxRow,maxRow,m->M,dirUp), errS))		goto err;
	#ifdef ROWLENS
	if(cudaErr( hipMalloc(&(dstLocal.RL),sizeof(*(dstLocal.RL)*dstLocal.M)), errS))			goto err;
	if(cudaErr( hipMemcpy(dstLocal.RL,m->RL,sizeof(*(dstLocal.RL)*dstLocal.M)), errS))			goto err;
	#endif
	//write dst-copy in destination CUDA mem struct
	if(cudaErr( hipMemcpy( dst,&dstLocal,sizeof(dstLocal),dirUp), errS))						goto err;
	return EXIT_SUCCESS;

	err:
	hipFree(dstLocal.JA);
	hipFree(dstLocal.AS);
	#ifdef ROWLENS
	hipFree(dstLocal.RL);
	#endif
	return EXIT_FAILURE;
}
